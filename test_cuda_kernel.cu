#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

// computes one projetion view
__global__ void projection_view_kernel(
                    const torch::PackedTensorAccessor32<float,2> image,
                    torch::PackedTensorAccessor32<float,2> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const int nbins){

  const int nx = image.size(0);
  const int ny = image.size(1);

  // get view index "sindex" from block/thread
  // const int n = blockIdx.y;
  // const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int sindex = threadIdx.x;

  auto s = sindex*ds;

  // location of the source
  auto xsource = radius*cos(s);
  auto ysource = radius*sin(s);

  // detector center
  auto xDetCenter = (radius - source_to_detector)*cos(s);
  auto yDetCenter = (radius - source_to_detector)*sin(s);

  // unit vector in the direction of the detector line
  auto eux = -sin(s);
  auto euy =  cos(s);

  //loop over detector views
  for (int uindex = 0; uindex < nbins; uindex++){
    auto u = u0 + (uindex+0.5)*du;
    auto xbin = xDetCenter + eux*u;
    auto ybin = yDetCenter + euy*u;

    auto xl = x0;
    auto yl = y0;

    auto xdiff = xbin-xsource;
    auto ydiff = ybin-ysource;
    auto xad = abs(xdiff)*dy;
    auto yad = abs(ydiff)*dx;

    float raysum = 0.0; // acculumator variable

    if (xad > yad){  // loop through x-layers of image if xad>yad. This ensures ray hits only one or two pixels per layer
      auto slope = ydiff/xdiff;
      auto travPixlen = dx*sqrt(1.0+slope*slope);
      auto yIntOld = ysource+slope*(xl-xsource);
      int iyOld = static_cast<int>(floor((yIntOld-y0)/dy));
      // loop over x-layers
      for (int ix = 0; ix < nx; ix++){
         auto x=xl+dx*(ix + 1.0);
         auto yIntercept=ysource+slope*(x-xsource);
         int iy = static_cast<int>(floor((yIntercept-y0)/dy));
         if (iy == iyOld){ // if true, ray stays in the same pixel for this x-layer
            if ((iy >= 0) && (iy < ny)) {
               raysum += travPixlen*image[ix][iy];
            }
         } else {    // else case is if ray hits two pixels for this x-layer
            auto yMid=dy*std::max(iy,iyOld)+yl;
            auto ydist1=abs(yMid-yIntOld);
            auto ydist2=abs(yIntercept-yMid);
            auto frac1=ydist1/(ydist1+ydist2);
            auto frac2=1.0-frac1;
            if ((iyOld >= 0) && (iyOld < ny)){
               raysum += frac1*travPixlen*image[ix][iyOld];
             }
            if ((iy>=0) && (iy<ny)){
               raysum += frac2*travPixlen*image[ix][iy];
             }
         }
         iyOld=iy;
         yIntOld=yIntercept;
       }

    } else {// through y-layers of image if xad<=yad
      auto slopeinv=xdiff/ydiff;
      auto travPixlen=dy*sqrt(1.0+slopeinv*slopeinv);
      auto xIntOld=xsource+slopeinv*(yl-ysource);
      int ixOld= static_cast<int>(floor((xIntOld-x0)/dx));
      // loop over y-layers
      for (int iy = 0; iy < ny; iy++){
         auto y=yl+dy*(iy + 1.0);
         auto xIntercept=xsource+slopeinv*(y-ysource);
         int ix = static_cast<int>(floor((xIntercept-x0)/dx));
         if (ix == ixOld){// if true, ray stays in the same pixel for this y-layer
            if ((ix >= 0) && (ix < nx)){
               raysum += travPixlen*image[ix][iy];
             }
         } else {  // else case is if ray hits two pixels for this y-layer
            auto xMid=dx*std::max(ix,ixOld)+xl;
            auto xdist1=abs(xMid-xIntOld);
            auto xdist2=abs(xIntercept-xMid);
            auto frac1=xdist1/(xdist1+xdist2);
            auto frac2=1.0-frac1;
            if ((ixOld >= 0) && (ixOld < nx)){
               raysum += frac1*travPixlen*image[ixOld][iy];
            }
            if ((ix>=0) && (ix<nx)){
               raysum += frac2*travPixlen*image[ix][iy];
            }
         }
         ixOld = ix;
         xIntOld = xIntercept;
       }
    }
    sinogram[sindex][uindex]=raysum;
 }
}

// computes backprojetion over one view
__global__ void backprojection_view_kernel(
                    torch::PackedTensorAccessor32<float,2> image,
                    const torch::PackedTensorAccessor32<float,2> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const int nbins,
                    const float fov_radius,
                    const float pi){

                    const int nx = image.size(0);
                    const int ny = image.size(1);

                    const int sindex = threadIdx.x;
                    float s = sindex*ds;

                    // location of the source
                    float xsource = radius*cos(s);
                    float ysource = radius*sin(s);

                    // detector center
                    float xDetCenter = (radius - source_to_detector)*cos(s);
                    float yDetCenter = (radius - source_to_detector)*sin(s);

                    // unit vector in the direction of the detector line
                    float eux = -sin(s);
                    float euy =  cos(s);

                    //Unit vector in the direction perpendicular to the detector line
                    float ewx = cos(s);
                    float ewy = sin(s);

                    for (int iy = 0; iy < ny; iy++){
                       float pix_y = y0 + dy*(iy+0.5);
                       for (int ix = 0; ix < nx; ix++){
                          float pix_x = x0 + dx*(ix+0.5);

                          float frad = sqrt(pix_x*pix_x + pix_y*pix_y);
                          float fphi = atan2(pix_y,pix_x);
                          if (frad<=fov_radius){
                             float bigu = (radius+frad*sin(s-fphi-pi/2.0))/radius;
                             float bpweight = 1.0/(bigu*bigu);

                             float ew_dot_source_pix = (pix_x-xsource)*ewx + (pix_y-ysource)*ewy;
                             float rayratio = -source_to_detector/ew_dot_source_pix;

                             float det_int_x = xsource+rayratio*(pix_x-xsource);
                             float det_int_y = ysource+rayratio*(pix_y-ysource);

                             float upos = ((det_int_x-xDetCenter)*eux +(det_int_y-yDetCenter)*euy);
                             float det_value;

                             if ((upos-u0 >= du/2.0) && (upos-u0 < detectorlength-du/2.0)){
                                float bin_loc = (upos-u0)/du + 0.5;
                                int nbin1 = static_cast<int>(bin_loc)-1;
                                int nbin2 = nbin1+1;
                                float frac= bin_loc - static_cast<int>(bin_loc);
                                det_value = frac*sinogram[sindex][nbin2]+(1.0-frac)*sinogram[sindex][nbin1];
                                atomicAdd(image[ix][iy],bpweight*det_value*ds);
                              }
                             // } else {
                             //    det_value = 0.0;
                             // }
                             // image[ix][iy] += bpweight*det_value*ds;
                         }
                      }
                   }

}

torch::Tensor circularFanbeamProjection_cuda(const torch::Tensor image, const int nx, const int ny, const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
    const float dx = ximageside/nx;
    const float dy = yimageside/ny;
    const float x0 = -ximageside/2.0;
    const float y0 = -yimageside/2.0;

    // compute length of detector so that it views the inscribed FOV of the image array
    const float fanangle2 = std::asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
    const float detectorlength = 2.0*std::tan(fanangle2)*source_to_detector;
    const float u0 = -detectorlength/2.0;

    const float du = detectorlength/nbins;
    const float ds = slen/nviews;

    const auto image_a = image.packed_accessor32<float,2>();

    const auto options = torch::TensorOptions().device(torch::kCUDA);
    torch::Tensor sinogram = torch::zeros({nviews, nbins}, options);
    auto sinogram_a = sinogram.packed_accessor32<float,2>();

    const int threads = 512; //one per view?
    // const dim3 blocks((512 + threads - 1) / threads, 1);
    const int blocks = 1; //match to batch size in future?

    projection_view_kernel<<<blocks, threads>>>(image_a,
                                                sinogram_a,
                                                dx,
                                                dy,
                                                x0,
                                                y0,
                                                fanangle2,
                                                detectorlength,
                                                u0,
                                                du,
                                                ds,
                                                radius,
                                                source_to_detector,
                                                nbins);

    return sinogram;
}

torch::Tensor circularFanbeamBackProjection_cuda(const torch::Tensor sinogram, const int nx, const int ny,
                              const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
   const float dx = ximageside/nx;
   const float dy = yimageside/ny;
   const float x0 = -ximageside/2.0;
   const float y0 = -yimageside/2.0;

   // compute length of detector so that it views the inscribed FOV of the image array
   const float fanangle2 = asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
   const float detectorlength = 2.0*tan(fanangle2)*source_to_detector;
   const float u0 = -detectorlength/2.0;

   const float du = detectorlength/nbins;
   const float ds = slen/nviews;

   const float fov_radius = ximageside/2.0;

   const auto options = torch::TensorOptions().device(torch::kCUDA);
   torch::Tensor image = torch::zeros({nx, ny}, options); //initialize image
   auto image_a = image.packed_accessor32<float,2>(); //accessor for updating values of image

   const auto sinogram_a = sinogram.packed_accessor32<float,2>(); //accessor for accessing values of sinogram

   const float pi = 4*atan(1);

   const int threads = 512; //one per view?
   // const dim3 blocks((512 + threads - 1) / threads, 1);
   const int blocks = 1; //match to batch size in future?

   backprojection_view_kernel<<<blocks, threads>>>(image_a,
                                                   sinogram_a,
                                                   dx,
                                                   dy,
                                                   x0,
                                                   y0,
                                                   fanangle2,
                                                   detectorlength,
                                                   u0,
                                                   du,
                                                   ds,
                                                   radius,
                                                   source_to_detector,
                                                   nbins,
                                                   fov_radius,
                                                   pi);
   return image;
}
