#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

inline int roundup_div(const int x, const int y) {
    return x / y + (x % y != 0);
}

// computes one projetion view
__global__ void projection_view_kernel(
                    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> image,
                    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const int nbins,
                    const int nviews,
                    const int nx,
                    const int ny){

  // const int nx = image.size(1);
  // const int ny = image.size(2);

  // const int ib = blockIdx.x;
  // const int sindex = threadIdx.x;

  const int uindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int sindex = blockIdx.y * blockDim.y + threadIdx.y;
  const int ib = blockIdx.z;

  //loop over detector views
  // for (int uindex = 0; uindex < nbins; uindex++){
  if ((uindex < nbins) && (sindex < nviews)) {
    auto s = sindex*ds;

    // location of the source
    auto xsource = radius*cos(s);
    auto ysource = radius*sin(s);

    // detector center
    auto xDetCenter = (radius - source_to_detector)*cos(s);
    auto yDetCenter = (radius - source_to_detector)*sin(s);

    // unit vector in the direction of the detector line
    auto eux = -sin(s);
    auto euy =  cos(s);

    auto u = u0 + (uindex+0.5)*du;
    auto xbin = xDetCenter + eux*u;
    auto ybin = yDetCenter + euy*u;

    auto xl = x0;
    auto yl = y0;

    auto xdiff = xbin-xsource;
    auto ydiff = ybin-ysource;
    auto xad = abs(xdiff)*dy;
    auto yad = abs(ydiff)*dx;

    float raysum = 0.0; // acculumator variable

    if (xad > yad){  // loop through x-layers of image if xad>yad. This ensures ray hits only one or two pixels per layer
      auto slope = ydiff/xdiff;
      auto travPixlen = dx*sqrt(1.0+slope*slope);
      auto yIntOld = ysource+slope*(xl-xsource);
      int iyOld = static_cast<int>(floor((yIntOld-y0)/dy));
      // loop over x-layers
      for (int ix = 0; ix < nx; ix++){
         auto x=xl+dx*(ix + 1.0);
         auto yIntercept=ysource+slope*(x-xsource);
         int iy = static_cast<int>(floor((yIntercept-y0)/dy));
         if (iy == iyOld){ // if true, ray stays in the same pixel for this x-layer
            if ((iy >= 0) && (iy < ny)) {
               raysum += travPixlen*image[ib][ix][iy];
            }
         } else {    // else case is if ray hits two pixels for this x-layer
            auto yMid=dy*max(iy,iyOld)+yl;
            auto ydist1=abs(yMid-yIntOld);
            auto ydist2=abs(yIntercept-yMid);
            auto frac1=ydist1/(ydist1+ydist2);
            auto frac2=1.0-frac1;
            if ((iyOld >= 0) && (iyOld < ny)){
               raysum += frac1*travPixlen*image[ib][ix][iyOld];
             }
            if ((iy>=0) && (iy<ny)){
               raysum += frac2*travPixlen*image[ib][ix][iy];
             }
         }
         iyOld=iy;
         yIntOld=yIntercept;
       }

    } else {// through y-layers of image if xad<=yad
      auto slopeinv=xdiff/ydiff;
      auto travPixlen=dy*sqrt(1.0+slopeinv*slopeinv);
      auto xIntOld=xsource+slopeinv*(yl-ysource);
      int ixOld= static_cast<int>(floor((xIntOld-x0)/dx));
      // loop over y-layers
      for (int iy = 0; iy < ny; iy++){
         auto y=yl+dy*(iy + 1.0);
         auto xIntercept=xsource+slopeinv*(y-ysource);
         int ix = static_cast<int>(floor((xIntercept-x0)/dx));
         if (ix == ixOld){// if true, ray stays in the same pixel for this y-layer
            if ((ix >= 0) && (ix < nx)){
               raysum += travPixlen*image[ib][ix][iy];
             }
         } else {  // else case is if ray hits two pixels for this y-layer
            auto xMid=dx*max(ix,ixOld)+xl;
            auto xdist1=abs(xMid-xIntOld);
            auto xdist2=abs(xIntercept-xMid);
            auto frac1=xdist1/(xdist1+xdist2);
            auto frac2=1.0-frac1;
            if ((ixOld >= 0) && (ixOld < nx)){
               raysum += frac1*travPixlen*image[ib][ixOld][iy];
            }
            if ((ix>=0) && (ix<nx)){
               raysum += frac2*travPixlen*image[ib][ix][iy];
            }
         }
         ixOld = ix;
         xIntOld = xIntercept;
       }
    }
    sinogram[ib][sindex][uindex]=raysum;
 }
}

// computes one backprojection view
__global__ void backprojection_view_kernel(
                    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> image,
                    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const float fov_radius,
                    const int nbins,
                    const int nviews,
                    const int nx,
                    const int ny){

  const int uindex = blockIdx.x * blockDim.x + threadIdx.x; //detector index
  const int sindex = blockIdx.y * blockDim.y + threadIdx.y; //view index
  const int ib = blockIdx.z; //batch index
  // const int ib = blockIdx.x;
  // const int sindex = threadIdx.x;

  //compute backprojection for a single ray
  if ((uindex < nbins) && (sindex < nviews)) {
    const auto sinoval = sinogram[ib][sindex][uindex];

    const float s = sindex*ds;

    // location of the source
    const float xsource = radius*cos(s);
    const float ysource = radius*sin(s);

    // detector center
    const float xDetCenter = (radius - source_to_detector)*cos(s);
    const float yDetCenter = (radius - source_to_detector)*sin(s);

    // unit vector in the direction of the detector line
    const float eux = -sin(s);
    const float euy =  cos(s);

    const float fov_radius2 = fov_radius*fov_radius; //used to set image mask

    float u = u0+(uindex+0.5)*du;
    float xbin = xDetCenter + eux*u;
    float ybin = yDetCenter + euy*u;

    float xl=x0;
    float yl=y0;

    float xdiff=xbin-xsource;
    float ydiff=ybin-ysource;
    float xad=abs(xdiff)*dy;
    float yad=abs(ydiff)*dx;

    if (xad>yad){   // loop through x-layers of image if xad>yad. This ensures ray hits only one or two pixels per layer
       float slope=ydiff/xdiff;
       float travPixlen=dx*sqrt(1.0+slope*slope);
       float yIntOld=ysource + slope*(xl-xsource);
       int iyOld = static_cast<int>(floor((yIntOld-y0)/dy));
       for (int ix = 0; ix < nx; ix++){
          float x = xl + dx*(ix + 1.0);
          float yIntercept=ysource+slope*(x-xsource);
          int iy = static_cast<int>(floor((yIntercept-y0)/dy));
          float pix_x = x0 + dx*(ix+0.5); //used to set mask
          float pix_y = y0 + dy*(iy+0.5); //used to set mask
          float pix_y_old = y0 + dy*(iyOld+0.5); // used to set mask
            if (iy == iyOld){ // if true, ray stays in the same pixel for this x-layer
             if ((pix_x*pix_x + pix_y*pix_y <= fov_radius2) && (iy >= 0) && (iy < ny)){
                atomicAdd(&image[ib][ix][iy],sinoval*travPixlen);
                // image[ib][ix][iy] += sinoval*travPixlen;
              }
          } else {    // else case is if ray hits two pixels for this x-layer
             float yMid = dy*max(iy,iyOld)+yl;
             float ydist1 = abs(yMid-yIntOld);
             float ydist2 = abs(yIntercept-yMid);
             float frac1 = ydist1/(ydist1+ydist2);
             float frac2 = 1.0-frac1;
             if ((iyOld >= 0) && (iyOld < ny) && (pix_x*pix_x + pix_y_old*pix_y_old <= fov_radius2)){
                atomicAdd(&image[ib][ix][iyOld],frac1*sinoval*travPixlen);
                // image[ib][ix][iyOld] += frac1*sinoval*travPixlen;
              }
             if ((iy >= 0) && (iy < ny) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)) {
                atomicAdd(&image[ib][ix][iy],frac2*sinoval*travPixlen);
                // image[ib][ix][iy] += frac2*sinoval*travPixlen;
              }
          }
          iyOld=iy;
          yIntOld=yIntercept;
        }
    } else { //loop through y-layers of image if xad<=yad
       float slopeinv=xdiff/ydiff;
       float travPixlen=dy*sqrt(1.0+slopeinv*slopeinv);
       float xIntOld=xsource+slopeinv*(yl-ysource);
       int ixOld = static_cast<int>(floor((xIntOld-x0)/dx));
       for (int iy = 0; iy < ny; iy++){
          float y = yl + dy*(iy + 1.0);
          float xIntercept = xsource+slopeinv*(y-ysource);
          int ix = static_cast<int>(floor((xIntercept-x0)/dx));
          float pix_x = x0 + dx*(ix+0.5);
          float pix_y = y0 + dy*(iy+0.5);
          float pix_x_old = x0 + dx*(ixOld+0.5); // used to set mask
          if (ix == ixOld){ // if true, ray stays in the same pixel for this y-layer
             if ((ix >= 0) && (ix < nx) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)) {
                atomicAdd(&image[ib][ix][iy],sinoval*travPixlen);
                // image[ib][ix][iy] += sinoval*travPixlen;
              }
          } else { // else case is if ray hits two pixels for this y-layer
             float xMid = dx*max(ix,ixOld)+xl;
             float xdist1 = abs(xMid-xIntOld);
             float xdist2 = abs(xIntercept-xMid);
             float frac1 = xdist1/(xdist1+xdist2);
             float frac2=1.0-frac1;
             if ((ixOld >= 0) && (ixOld < nx) && (pix_x_old*pix_x_old + pix_y*pix_y <= fov_radius2)){
                atomicAdd(&image[ib][ixOld][iy],frac1*sinoval*travPixlen);
                // image[ib][ixOld][iy] += frac1*sinoval*travPixlen;
              }
             if ((ix >= 0) && (ix < nx) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)){
                atomicAdd(&image[ib][ix][iy],frac2*sinoval*travPixlen);
                // image[ib][ix][iy] += frac2*sinoval*travPixlen;
              }
          }
          ixOld = ix;
          xIntOld = xIntercept;
       }
     }
   } // end uindex for loop

}

// computes pixel-driven backprojetion over one view
__global__ void backprojection_pix_view_kernel(
                    torch::PackedTensorAccessor32<float,3> image,
                    const torch::PackedTensorAccessor32<float,3> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const int nbins,
                    const float fov_radius,
                    const float pi){

                    const int nx = image.size(1);
                    const int ny = image.size(2);

                    const int sindex = threadIdx.x;
                    const int ib = blockIdx.x;

                    const float s = sindex*ds;

                    // location of the source
                    const float xsource = radius*cos(s);
                    const float ysource = radius*sin(s);

                    // detector center
                    const float xDetCenter = (radius - source_to_detector)*cos(s);
                    const float yDetCenter = (radius - source_to_detector)*sin(s);

                    // unit vector in the direction of the detector line
                    const float eux = -sin(s);
                    const float euy =  cos(s);

                    //Unit vector in the direction perpendicular to the detector line
                    const float ewx = cos(s);
                    const float ewy = sin(s);

                    for (int iy = 0; iy < ny; iy++){
                       float pix_y = y0 + dy*(iy+0.5);
                       for (int ix = 0; ix < nx; ix++){
                          float pix_x = x0 + dx*(ix+0.5);

                          float frad = sqrt(pix_x*pix_x + pix_y*pix_y);
                          float fphi = atan2(pix_y,pix_x);
                          if (frad<=fov_radius){
                             float bigu = (radius+frad*sin(s-fphi-pi/2.0))/radius;
                             float bpweight = 1.0/(bigu*bigu);

                             float ew_dot_source_pix = (pix_x-xsource)*ewx + (pix_y-ysource)*ewy;
                             float rayratio = -source_to_detector/ew_dot_source_pix;

                             float det_int_x = xsource+rayratio*(pix_x-xsource);
                             float det_int_y = ysource+rayratio*(pix_y-ysource);

                             float upos = ((det_int_x-xDetCenter)*eux +(det_int_y-yDetCenter)*euy);
                             float det_value;

                             if ((upos-u0 >= du/2.0) && (upos-u0 < detectorlength-du/2.0)){
                                float bin_loc = (upos-u0)/du + 0.5;
                                int nbin1 = static_cast<int>(bin_loc)-1;
                                int nbin2 = nbin1+1;
                                float frac= bin_loc - static_cast<int>(bin_loc);
                                det_value = frac*sinogram[ib][sindex][nbin2]+(1.0-frac)*sinogram[ib][sindex][nbin1];
                                atomicAdd(&image[ib][ix][iy],bpweight*det_value*ds);
                              }
                             // } else {
                             //    det_value = 0.0;
                             // }
                             // image[ix][iy] += bpweight*det_value*ds;
                         }
                      }
                   }

}

torch::Tensor circularFanbeamProjection_cuda(const torch::Tensor image, const int nx, const int ny, const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
    const float dx = ximageside/nx;
    const float dy = yimageside/ny;
    const float x0 = -ximageside/2.0;
    const float y0 = -yimageside/2.0;

    // compute length of detector so that it views the inscribed FOV of the image array
    const float fanangle2 = std::asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
    const float detectorlength = 2.0*std::tan(fanangle2)*source_to_detector;
    const float u0 = -detectorlength/2.0;

    const float du = detectorlength/nbins;
    const float ds = slen/nviews;

    const auto image_a = image.packed_accessor32<float,3,torch::RestrictPtrTraits>();
    const int batch_size = image_a.size(0); //batch_size

    const auto options = torch::TensorOptions().dtype(image.dtype()).device(image.device());
    auto sinogram = torch::zeros({batch_size, nviews, nbins}, options);
    auto sinogram_a = sinogram.packed_accessor32<float,3,torch::RestrictPtrTraits>();

    // parallize over rays, batches
    dim3 block_dim(16, 16);
    dim3 grid_dim(roundup_div(nbins, 16), roundup_div(nviews, 16), batch_size );

    projection_view_kernel<<<grid_dim, block_dim>>>(image_a,
                                                sinogram_a,
                                                dx,
                                                dy,
                                                x0,
                                                y0,
                                                fanangle2,
                                                detectorlength,
                                                u0,
                                                du,
                                                ds,
                                                radius,
                                                source_to_detector,
                                                nbins,
                                                nviews,
                                                nx,
                                                ny);

    return sinogram;
}

// exact matrix transpose of circularFanbeamProjection
torch::Tensor circularFanbeamBackProjection_cuda(const torch::Tensor sinogram, const int nx, const int ny,
                              const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
   const float dx = ximageside/nx;
   const float dy = yimageside/ny;
   const float x0 = -ximageside/2.0;
   const float y0 = -yimageside/2.0;

   // compute length of detector so that it views the inscribed FOV of the image array
   const float fanangle2 = std::asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
   const float detectorlength = 2.0*std::tan(fanangle2)*source_to_detector;
   const float u0 = -detectorlength/2.0;

   const float du = detectorlength/nbins;
   const float ds = slen/nviews;

   const float fov_radius = ximageside/2.0;

   const auto sinogram_a = sinogram.packed_accessor32<float,3,torch::RestrictPtrTraits>();
   const int batch_size = sinogram_a.size(0); //batch_size

   const auto options = torch::TensorOptions().dtype(sinogram.dtype()).device(sinogram.device());
   auto image = torch::zeros({batch_size, nx, ny}, options);
   auto image_a = image.packed_accessor32<float,3,torch::RestrictPtrTraits>();

   // const int threads = nviews; //one per view, max 1024 -- todo: add input validation
   // const int blocks = batch_size; //match to batch size

   // parallize over rays, batches
   dim3 block_dim(16, 16);
   dim3 grid_dim(roundup_div(nbins, 16), roundup_div(nviews, 16), batch_size );

   backprojection_view_kernel<<<grid_dim, block_dim>>>(image_a,
                                               sinogram_a,
                                               dx,
                                               dy,
                                               x0,
                                               y0,
                                               fanangle2,
                                               detectorlength,
                                               u0,
                                               du,
                                               ds,
                                               radius,
                                               source_to_detector,
                                               fov_radius,
                                               nbins,
                                               nviews,
                                               nx,
                                               ny);

    return image;
}


torch::Tensor circularFanbeamBackProjectionPixelDriven_cuda(const torch::Tensor sinogram, const int nx, const int ny,
                              const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
   const int batch_size = sinogram.size(0);
   const float dx = ximageside/nx;
   const float dy = yimageside/ny;
   const float x0 = -ximageside/2.0;
   const float y0 = -yimageside/2.0;

   // compute length of detector so that it views the inscribed FOV of the image array
   const float fanangle2 = asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
   const float detectorlength = 2.0*tan(fanangle2)*source_to_detector;
   const float u0 = -detectorlength/2.0;

   const float du = detectorlength/nbins;
   const float ds = slen/nviews;

   const float fov_radius = ximageside/2.0;

   const auto options = torch::TensorOptions().device(torch::kCUDA);
   torch::Tensor image = torch::zeros({batch_size, nx, ny}, options); //initialize image
   auto image_a = image.packed_accessor32<float,3>(); //accessor for updating values of image

   const auto sinogram_a = sinogram.packed_accessor32<float,3>(); //accessor for accessing values of sinogram

   const float pi = 4*atan(1);

   const int threads = nviews; //one per view, max 1024
   const int blocks = batch_size; //match to batch size

   backprojection_pix_view_kernel<<<blocks, threads>>>(image_a,
                                                   sinogram_a,
                                                   dx,
                                                   dy,
                                                   x0,
                                                   y0,
                                                   fanangle2,
                                                   detectorlength,
                                                   u0,
                                                   du,
                                                   ds,
                                                   radius,
                                                   source_to_detector,
                                                   nbins,
                                                   fov_radius,
                                                   pi);
   return image;
}
